#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates how use texture fetches in CUDA
 *
 * This sample takes an input PGM image (image_filename) and generates
 * an output PGM image (image_filename_out).  This CUDA kernel performs
 * a simple 2D transform (rotation) on the texture coordinates (u,v).
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

#include <opencv2/imgproc/imgproc_c.h>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

#define SCALED_WIDTH 64
#define SCALED_HEIGHT 64

#define SCALES 33


// Define the files that are to be save and the reference images for validation
const char *imageFilename = "lena_bw.pgm";
const char *refFilename   = "ref_rotated.pgm";

const char *sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Constants
//const float scale = 0.8f;

__constant__ float d_scale_factors[512];
__constant__ uint8_t d_scales;

// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
/*__global__ void transformKernel(float *outputData,
                                int width,
                                int height
                                )
{
	int scaled_width = SCALED_WIDTH;
	int scaled_height = SCALED_HEIGHT;

    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	outputData[y * scaled_width + x] = tex2D(tex,
					(float) x / scaled_width, (float) y / scaled_height);

    // read from texture and write to global memory
//	for (int i = x; i < scaled_width; i += gridDim.x * blockDim.x) {
//		for (int j = y; j < scaled_height; j += gridDim.y * blockDim.y) {
//			outputData[j * scaled_width + i] = tex2D(tex,
//					(float) i / scaled_width, (float) j / scaled_height);
//		}
//	}
}*/

/*__global__ void transformKernel(float *outputData,
                                int width,
                                int height
                                )
{
	int scaled_width = SCALED_WIDTH;
	int scaled_height = SCALED_HEIGHT;
	float upper = 0.53;
	float lower = 0.48;

    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    outputData[y * scaled_width + x] = tex2D(tex,
    					(float) x * (upper-lower) / scaled_width +lower, (float) y * (upper-lower) / scaled_height +lower);

}*/

__global__ void transformKernel(float *outputData,
                                int width,
                                int height
                                )
{
	int scaled_width = SCALED_WIDTH;
	int scaled_height = SCALED_HEIGHT;
	float upper = (d_scale_factors[0] + 1) * 0.5;
	float lower = (1 - d_scale_factors[0]) * 0.5;

    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    outputData[y * scaled_width + x] = tex2D(tex,
    					(float) x * (upper-lower) / scaled_width +lower, (float) y * (upper-lower) / scaled_height +lower);

}
////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("%s starting...\n", sampleName);

    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "input",
                                     (char **) &imageFilename);

            if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
            {
                getCmdLineArgumentString(argc,
                                         (const char **) argv,
                                         "reference",
                                         (char **) &refFilename);
            }
            else
            {
                printf("-input flag should be used with -reference flag");
                exit(EXIT_FAILURE);
            }
        }
        else if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
        {
            printf("-reference flag should be used with -input flag");
            exit(EXIT_FAILURE);
        }
    }

    runTest(argc, argv);

    printf("%s completed, returned %s\n",
           sampleName,
           testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    int devID = findCudaDevice(argc, (const char **) argv);

    // load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    unsigned int scaled_width = SCALED_WIDTH; //floor(scale * width);
    unsigned int scaled_height = SCALED_HEIGHT; //floor(scale * height);
    unsigned int scaled_width_size = scaled_width * sizeof(float);
    unsigned int scaled_height_size = scaled_height * sizeof(float);

    float scale_step = 1.02;
    uint8_t scales = SCALES;
    float scale_factors[scales];

    for (int i=0; i<scales ; i++) {
    	scale_factors[i] = pow(scale_step, i+1);
    }

    for (int i=0; i<scales ; i++) {
    	scale_factors[i] = scale_factors[i] / scale_factors[scales-1];
    }

    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //Load reference image from image (output)
    float *hDataRef = (float *) malloc(size);
    char *refPath = sdkFindFilePath(refFilename, argv[0]);

    if (refPath == NULL)
    {
        printf("Unable to find reference image file: %s\n", refFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(refPath, &hDataRef, &width, &height);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_scale_factors), scale_factors, scales*sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_scales), &scales, sizeof(uint8_t)));

    // Allocate device memory for result
    float *dData = NULL;
    checkCudaErrors(hipMalloc((void **) &dData, scaled_width_size * scaled_height_size));

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,
                                    &channelDesc,
                                    width,
                                    height));
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      hData,
                                      size,
                                      hipMemcpyHostToDevice));

    // Set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(scaled_width / dimBlock.x, scaled_height / dimBlock.y, 1);

    // Warmup
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height);

    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",
           (width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(scaled_width_size * scaled_height_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData,
                               dData,
                               scaled_width_size * scaled_height_size,
                               hipMemcpyDeviceToHost));

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
    sdkSavePGM(outputFilename, hOutputData, scaled_width, scaled_height);
    printf("Wrote '%s'\n", outputFilename);

//    // Write regression file if necessary
//    if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
//    {
//        // Write file for regression test
//        sdkWriteFile<float>("./data/regression.dat",
//                            hOutputData,
//                            width*height,
//                            0.0f,
//                            false);
//    }
//    else
//    {
//        // We need to reload the data from disk,
//        // because it is inverted upon output
//        sdkLoadPGM(outputFilename, &hOutputData, &width, &height);
//
//        printf("Comparing files\n");
//        printf("\toutput:    <%s>\n", outputFilename);
//        printf("\treference: <%s>\n", refPath);
//
//        testResult = compareData(hOutputData,
//                                 hDataRef,
//                                 width*height,
//                                 MAX_EPSILON_ERROR,
//                                 0.15f);
//    }

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
    free(imagePath);
    free(refPath);
}
